#include "hip/hip_runtime.h"
#include "bfc.cuh"
#include <hip/hip_runtime.h>

/**
 * butterfly counting with vertex priority
 * @param n number of vertices
 * @param blk_num block number
 * @param cnt the number of butterfly counting
 */
__global__ auto bfc_kernel(uint n, const uint* d_offset, const uint* d_neighbors, uint* d_bitmaps, int* last_uses,
                           const uint* d_degree, uint const blk_num, ull* cnt) -> void {
    uint bid = blockIdx.x;

    __shared__ uint* d_bitmap;
    __shared__ int* last_use;
    __shared__ uint last_idx;
    __shared__ ull local_cnt;

    if (threadIdx.x == 0) {
        d_bitmap = d_bitmaps + bid * n;
        last_use = last_uses + bid * n;
        last_idx = 0;
        local_cnt = 0;
    }

    __syncthreads();

    for (uint u = bid; u < n; u += blk_num) {

        if (u >= n) break;

        __syncthreads();

        for (uint j = threadIdx.x; j < last_idx; j += blockDim.x) {
            if (j >= last_idx) break;
            d_bitmap[last_use[j]] = 0;
        }
        __syncthreads();

        if (threadIdx.x == 0) {
            last_idx = 0;
            local_cnt = 0;
        }

        __syncthreads();

        uint const u_nbr_len = d_degree[u];
        uint const* u_nbr = d_neighbors + d_offset[u];

        for (uint i = threadIdx.x; i < u_nbr_len; i += blockDim.x) {
            if (i >= u_nbr_len) break;

            uint v = u_nbr[i];

            if (d_degree[u] < d_degree[v]) continue;
            if (d_degree[u] == d_degree[v] && u <= v) continue;

            uint const* v_nbr = d_neighbors + d_offset[v];
            uint const v_nbr_len = d_degree[v];

            for (uint j = 0; j < v_nbr_len; j++) {
                uint w = v_nbr[j];

                if (d_degree[u] < d_degree[w]) continue;
                if (d_degree[u] == d_degree[w] && u <= w) continue;


                uint old_val = atomicAdd(d_bitmap + w, 1);
                atomicAdd(&local_cnt, old_val);

                if (old_val == 0) {
                    uint idxx = atomicAdd(&last_idx, 1);
                    last_use[idxx] = w;
                }
            }
        }

        __syncthreads();
         if (threadIdx.x == 0) {
             atomicAdd(cnt, local_cnt);
         }

    }
}

/**
 * counting butterflies for each edge with vertex priority
 * @param n number of vertices
 * @param blk_num block number
 */
__global__ auto ebfc_kernel(uint n, const uint* d_offset, const uint* d_neighbors,
                            const uint* d_edge_ids, uint* d_edge_support,
                            uint* d_bitmaps, int* last_uses, const uint* d_degree, uint const blk_num) -> void {

    uint const bid = blockIdx.x;

    __shared__ uint* d_bitmap;
    __shared__ int* last_use;
    __shared__ uint last_idx;

    if (threadIdx.x == 0) {
        d_bitmap = d_bitmaps + bid * n;
        last_use = last_uses + bid * n;
        last_idx = 0;
    }

    __syncthreads();

    for (uint u = bid; u < n; u += blk_num) {

        if (u >= n) break;

        __syncthreads();

        for (uint j = threadIdx.x; j < last_idx; j += blockDim.x) {
            if (j >= last_idx) break;
            d_bitmap[last_use[j]] = 0;
        }
        __syncthreads();

        if (threadIdx.x == 0) last_idx = 0;

        __syncthreads();

        uint const u_nbr_len = d_degree[u];
        uint const* u_nbr = d_neighbors + d_offset[u];

        for (uint i = threadIdx.x; i < u_nbr_len; i += blockDim.x) {
            if (i >= u_nbr_len) break;

            uint v = u_nbr[i];

            if (d_degree[u] < d_degree[v]) continue;
            if (d_degree[u] == d_degree[v] && u <= v) continue;

            uint const* v_nbr = d_neighbors + d_offset[v];
            uint const v_nbr_len = d_degree[v];

            for (uint j = 0; j < v_nbr_len; j++) {
                uint w = v_nbr[j];

                if (d_degree[u] < d_degree[w]) continue;
                if (d_degree[u] == d_degree[w] && u <= w) continue;

                uint old_val = atomicAdd(d_bitmap + w, 1);

                if (old_val == 0) {
                    uint idxx = atomicAdd(&last_idx, 1);
                    last_use[idxx] = w;
                }
            }
        }

        __syncthreads();

        for (uint i = threadIdx.x; i < u_nbr_len; i += blockDim.x) {
            if (i >= u_nbr_len) break;

            uint v = u_nbr[i];

            if (d_degree[u] < d_degree[v]) continue;
            if (d_degree[u] == d_degree[v] && u <= v) continue;

            uint const* v_nbr = d_neighbors + d_offset[v];
            uint const v_nbr_len = d_degree[v];

            for (auto j = 0; j < v_nbr_len; j++) {
                uint w = v_nbr[j];

                if (d_degree[u] < d_degree[w]) continue;
                if (d_degree[u] == d_degree[w] && u <= w) continue;

                if (d_bitmap[w] == 0) continue;

                int dlt = int(d_bitmap[w]) - 1;
                if (dlt) {
                    uint const uv = d_edge_ids[d_offset[u] + i];
                    uint const vw = d_edge_ids[d_offset[v] + j];

                    atomicAdd(d_edge_support + uv, dlt);
                    atomicAdd(d_edge_support + vw, dlt);
                }
            }
        }
    }
}

/**
 * butterfly counting with vertex priority
 * we used
 * @param g graph object
 */
auto bfc_evpp(Graph* g) -> void {

    uint* d_offset;
    uint* d_neighbors;
    uint* d_edge_support;
    uint* d_edge_ids;
    uint* d_bitmaps;
    int* d_last_uses;
    uint* d_degree;
    size_t free_memory;


    // alloca memory
    CER(hipMalloc(&d_offset, sizeof(uint) * (g->n + 1)));
    CER(hipMalloc(&d_neighbors, sizeof(uint) * g->m * 2));
    CER(hipMalloc(&d_edge_support, sizeof(uint) * g->m));
    CER(hipMalloc(&d_edge_ids, sizeof(uint) * g->m * 2));
    CER(hipMalloc(&d_degree, sizeof(uint) * g->n));

    hipMemGetInfo(&free_memory, nullptr);
    uint blk_num = free_memory  * 0.96 / (g->n * 4 * 2);
    blk_num = blk_num > g->n ? BLK_NUMS : blk_num;

    CER(hipMalloc(&d_bitmaps, sizeof(uint) * g->n * blk_num));
    CER(hipMalloc(&d_last_uses, sizeof(int) * g->n * blk_num));


    // copy memory from host to device
    hipMemcpy((void*) d_offset, (void*) g->offsets, sizeof(uint) * (g->n + 1), hipMemcpyHostToDevice);
    hipMemcpy((void*) d_neighbors, (void*) g->neighbors, sizeof(uint) * g->m * 2, hipMemcpyHostToDevice);
    hipMemset((void*) d_edge_support, 0, sizeof(uint) * g->m);
    hipMemcpy((void*) d_edge_ids, (void*) g->edge_ids, sizeof(uint) * g->m * 2, hipMemcpyHostToDevice);
    hipMemset((void*) d_bitmaps, 0, sizeof(uint) * g->n * blk_num);
    hipMemset((void*) d_last_uses, -1, sizeof(int) * g->n * blk_num);
    hipMemcpy((void*) d_degree, (void*) g->degrees, sizeof(uint) * g->n, hipMemcpyHostToDevice);

    ebfc_kernel<<<blk_num, BLK_DIM>>>(g->n, d_offset, d_neighbors, d_edge_ids, d_edge_support, d_bitmaps, d_last_uses, d_degree, blk_num);

    hipDeviceSynchronize();

    // get the max edge support
    hipMemcpy((void*) g->edge_support, (void*) d_edge_support, sizeof(uint) * g->m, hipMemcpyDeviceToHost);
    g->support_max = *std::max_element(g->edge_support, g->edge_support + g->m);
    log_info("butterfly counting with vertex priority on gpu with %d blocks, and the max edge support is %'d", blk_num, g->support_max);

// count butterfly
#ifdef COUNT_BUTTERFLY
    ull* d_cnt;
    CER(hipMalloc(&d_cnt, sizeof(ull)));
    CER(hipMemset(d_cnt, 0, sizeof(ull)));

    bfc_kernel<<<blk_num, BLK_DIM>>>(g->n, d_offset, d_neighbors, d_bitmaps, d_last_uses, d_degree, blk_num, d_cnt);
    hipDeviceSynchronize();

    ull cnt;
    hipMemcpy(&cnt, d_cnt, sizeof(ull), hipMemcpyDeviceToHost);
    log_info("total butterfly counting with vertex priority on gpu: %'llu", cnt);

    hipFree(d_cnt);
#endif

    // get synchronize error
    hipError_t cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        std::cerr << "CUDA Error: " << hipGetErrorString(cudaStatus) << std::endl;
        exit(EXIT_FAILURE);
    }

    // free cuda memory
    hipFree(d_offset);
    hipFree(d_neighbors);
    hipFree(d_edge_support);
    hipFree(d_edge_ids);
    hipFree(d_bitmaps);
    hipFree(d_last_uses);
    hipFree(d_degree);
}


/**
 * butterfly counting (cnt_only) with vertex priority
 * we used
 * @param g graph object
 */
auto bfc_vpp(Graph* g) -> void {

    uint* d_offset;
    uint* d_neighbors;
    uint* d_edge_support;
    uint* d_edge_ids;
    uint* d_bitmaps;
    int* d_last_uses;
    uint* d_degree;
    size_t free_memory;
    ull* d_cnt;


    // alloca memory
    CER(hipMalloc(&d_offset, sizeof(uint) * (g->n + 1)));
    CER(hipMalloc(&d_neighbors, sizeof(uint) * g->m * 2));
    CER(hipMalloc(&d_edge_support, sizeof(uint) * g->m));
    CER(hipMalloc(&d_edge_ids, sizeof(uint) * g->m * 2));
    CER(hipMalloc(&d_degree, sizeof(uint) * g->n));
    CER(hipMalloc(&d_cnt, sizeof(ull)));

    hipMemGetInfo(&free_memory, nullptr);
    uint blk_num = free_memory  * 0.96 / (g->n * 4 * 2);
    blk_num = blk_num > g->n ? BLK_NUMS : blk_num;

    CER(hipMalloc(&d_bitmaps, sizeof(uint) * g->n * blk_num));
    CER(hipMalloc(&d_last_uses, sizeof(int) * g->n * blk_num));


    // copy memory from host to device
    hipMemcpy((void*) d_offset, (void*) g->offsets, sizeof(uint) * (g->n + 1), hipMemcpyHostToDevice);
    hipMemcpy((void*) d_neighbors, (void*) g->neighbors, sizeof(uint) * g->m * 2, hipMemcpyHostToDevice);
    hipMemset((void*) d_edge_support, 0, sizeof(uint) * g->m);
    hipMemcpy((void*) d_edge_ids, (void*) g->edge_ids, sizeof(uint) * g->m * 2, hipMemcpyHostToDevice);
    hipMemset((void*) d_bitmaps, 0, sizeof(uint) * g->n * blk_num);
    hipMemset((void*) d_last_uses, -1, sizeof(int) * g->n * blk_num);
    hipMemcpy((void*) d_degree, (void*) g->degrees, sizeof(uint) * g->n, hipMemcpyHostToDevice);
    CER(hipMemset(d_cnt, 0, sizeof(ull)));

    bfc_kernel<<<blk_num, BLK_DIM>>>(g->n, d_offset, d_neighbors, d_bitmaps, d_last_uses, d_degree, blk_num, d_cnt);
    hipDeviceSynchronize();

    // get the max edge support
    ull cnt;
    hipMemcpy(&cnt, d_cnt, sizeof(ull), hipMemcpyDeviceToHost);
    log_info("butterfly counting with vertex priority on gpu with %d blocks, and btf number: %'llu", blk_num, cnt);


    // get synchronize error
    hipError_t cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        std::cerr << "CUDA Error: " << hipGetErrorString(cudaStatus) << std::endl;
        exit(EXIT_FAILURE);
    }

    // free cuda memory
    hipFree(d_offset);
    hipFree(d_neighbors);
    hipFree(d_edge_support);
    hipFree(d_edge_ids);
    hipFree(d_bitmaps);
    hipFree(d_last_uses);
    hipFree(d_degree);
    hipFree(d_cnt);
}
